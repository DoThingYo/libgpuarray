#include "hip/hip_runtime.h"
#include <sys/param.h>
#include <sys/stat.h>
#include <sys/uio.h>

#include <assert.h>
#include <fcntl.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "compyte_buffer.h"
#include "compyte_util.h"

#define CNDA_THREAD_SYNC hipDeviceSynchronize()

struct _gpudata {
    char *ptr;
};

struct _gpukernel {
    hipModule_t m;
    hipFunction_t k;
    void **args;
    unsigned int argcount;
};

static gpudata *cuda_alloc(void *ctx /* IGNORED */, size_t size)
{
    /* ctx is ignored since it is implied from the context stack */
    hipError_t err;
    gpudata *res;
    res = (gpudata *)malloc(sizeof(*res));
    if (res == NULL) {
        return NULL;
    }
    err = hipMalloc(&res->ptr, size);
    if (hipSuccess != err) {
        return NULL;
    }
    return res;
}

static void cuda_free(gpudata *d) {
    hipFree(d->ptr);
    free(d);
}

static int cuda_move(gpudata *dst, gpudata *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst->ptr, src->ptr, sz, hipMemcpyDeviceToDevice);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_read(void *dst, gpudata *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst, src->ptr, sz, hipMemcpyDeviceToHost);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_write(gpudata *dst, void *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst->ptr, src, sz, hipMemcpyHostToDevice);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_memset(gpudata *dst, int data, size_t bytes)
{
    hipError_t err;
    err = hipMemset(dst->ptr, data, bytes);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_offset(gpudata *buf, int off) {
    buf->ptr += off;
    return 0;
}

/* This is a unix version, might need a windows one. */
static int call_compiler(char *fname, char *oname) {
    int err;
    pid_t p;
    
    p = fork();
    if (p == 0) {
        /* Will need some way to specify arch (or detect it live) */
        execlp("nvcc", "-xcu", "--cubin", fname, "-o", oname, NULL);
        exit(1);
    } else if (p == -1) {
        return GA_SYS_ERROR;
    }
    if (waitpid(p, &err, 0) == -1)
        return GA_SYS_ERROR;
    if (WIFSIGNALED(err) || WEXITSTATUS(err) != 0) return GA_SYS_ERROR;
    return 0;
}

static gpukernel *cuda_newkernel(void *ctx /* IGNORED */, unsigned int count,
                                 const char **strings, const size_t *lengths,
                                 const char *fname) {
    char namebuf[MAXPATHLEN];
    char outbuf[MAXPATHLEN];
    char *tmpdir;
    int fd, err;
    ssize_t s;
    struct iovec descr[count];
    gpukernel *res;

    if (count == 0) return NULL;
    
    if (lengths == NULL) {
        for (unsigned int i = 0; i < count; i++) {
            descr[i].iov_base = (void *)strings[i];
            descr[i].iov_len = strlen(strings[i]);
        }
    } else {
        for (unsigned int i = 0; i < count; i++) {
            descr[i].iov_base = (void *)strings[i];
            descr[i].iov_len = lengths[i];
        }
    }
    
    tmpdir = getenv("TMPDIR");
    if (tmpdir == NULL) tmpdir = "/tmp";
    
    strlcpy(namebuf, tmpdir, sizeof(namebuf));
    strlcat(namebuf, "/compyte.cuda.XXXXXXXX", sizeof(namebuf));

    fd = mkstemp(namebuf);
    if (fd == -1) return NULL;
    
    strlcpy(outbuf, namebuf, sizeof(outbuf));
    strlcat(outbuf, ".cubin", sizeof(outbuf));
    
    s = writev(fd, descr, count);
    /* fd is not non-blocking so should have complete write */
    if (s == -1) {
        close(fd);
        unlink(namebuf);
        return NULL;
    }
    err = call_compiler(namebuf, outbuf);

    close(fd);
    unlink(namebuf);

    if (err != GA_NO_ERROR) return NULL;
        
    res = (gpukernel *)malloc(sizeof(*res));
    if (res == NULL) return NULL;
    res->args = NULL;
    res->argcount = 0;
    
    if (hipModuleLoad(&res->m, outbuf) != hipSuccess) {
        free(res);
        return NULL;
    }

    if (hipModuleGetFunction(&res->k, res->m, fname) != hipSuccess) {
        hipModuleUnload(res->m);
        free(res);
        return NULL;
    }

    return res;
}

static void cuda_freekernel(gpukernel *k) {
    for (unsigned int i = 0; i < k->argcount; i++)
        free(k->args[i]);
    free(k->args);
    hipModuleUnload(k->m);
    free(k);
}

static int cuda_setkernelarg(gpukernel *k, unsigned int index, size_t sz,
                             const void *val) {
    void *tmp;
    if (index > k->argcount) {
        tmp = calloc(index+1, sizeof(void *));
        if (tmp == NULL) return GA_MEMORY_ERROR;
        bcopy(k->args, tmp, sizeof(void *)*k->argcount);
        free(k->args);
        k->args = (void **)tmp;
        k->argcount = index+1;
    }
    tmp = malloc(sz);
    if (tmp == NULL) return GA_MEMORY_ERROR;
    bcopy(val, tmp, sz);
    k->args[index] = tmp;
    return GA_NO_ERROR;
}

static int cuda_setkernelargbuf(gpukernel *k, unsigned int index, gpudata *b) {
    return cuda_setkernelarg(k, index, sizeof(void *), &b->ptr);
}

static int cuda_callkernel(gpukernel *k, unsigned int gx, unsigned int gy,
                           unsigned int gz, unsigned int bx, unsigned int by,
                           unsigned int bz) {
    hipError_t err;
    
    err = hipModuleLaunchKernel(k->k, gx, gy, gz, bx, by, bz, 0, NULL, k->args, NULL);
    if (err != hipSuccess) {
        return GA_IMPL_ERROR;
    }
    err = hipCtxSynchronize();
    if (err != hipSuccess) {
        return GA_IMPL_ERROR;
    }
    return GA_NO_ERROR;
}

static const char ELEM_HEADER[] = "#define DTYPEA %s\n"
    "#define DTYPEB %s\n"
    "__global__ void elemk(const DTYPEA *a_data, DTYPEB *b_data) {"
    "const int idx = blockIdx.x * blockDim.x + threadIdx.x;"
    "const int numThreads = blockDim.x * gridDim.x;"
    "for (int i = idx; i < %zu; i += numThreads) {"
    "const DTYPEA *a = a_data;"
    "DTYPEB *b = b_data;";

static const char ELEM_FOOTER[] = "}}\n";

static inline unsigned int xmin(unsigned long a, unsigned long b) {
    return (unsigned int)((a < b) ? a : b);
}

static int cuda_elemwise(gpudata *input, gpudata *output, int intype,
                         int outtype, const char *op, unsigned int a_nd,
                         const size_t *a_dims, const ssize_t *a_str,
                         unsigned int b_nd, const size_t *b_dims,
                         const ssize_t *b_str) {
    char *strs[64];
    unsigned int count = 0;
    int res = GA_SYS_ERROR;
    
    size_t nEls = 1;
    gpukernel *k;

    for (unsigned int i = 0; i < a_nd; i++) {
        nEls *= a_dims[i];
    }
    
    if (asprintf(&strs[count], ELEM_HEADER,
                 compyte_get_type(intype)->cuda_name,
                 compyte_get_type(outtype)->cuda_name,
                 nEls) == -1)
        goto fail;
    count++;

    if (0) { /* contiguous case */
        if (asprintf(&strs[count], "b[i] %s a[i];", op) == -1)
            goto fail;
        count++;
    } else {
        if (compyte_elem_perdim(strs, &count, a_nd, a_dims, a_str, "a") == -1)
            goto fail;
        if (compyte_elem_perdim(strs, &count, b_nd, b_dims, b_str, "b") == -1)
            goto fail;

        if (asprintf(&strs[count], "b[0] %s a[0];", op) == -1)
            goto fail;
        count++;
    }

    strs[count] = (char *)ELEM_FOOTER;
    count++;
    
    assert(count < (sizeof(strs)/sizeof(strs[0])));

    res = GA_IMPL_ERROR;
    k = cuda_newkernel(NULL, count, (const char **)strs, NULL, "elemk");
    if (k == NULL) goto fail;
    res = cuda_setkernelargbuf(k, 0, input);
    if (res != GA_NO_ERROR) goto fail;
    res = cuda_setkernelargbuf(k, 1, output);
    if (res != GA_NO_ERROR) goto fail;

    /* XXX: Revise this crappy block/grid assigment */
    unsigned int gx, bx;
    bx = xmin(32, nEls);
    gx = xmin((nEls/bx)+((nEls % bx != 0)?1:0), 60);
    if (bx*gx < nEls)
        bx = xmin(nEls/gx, 512);

    res = cuda_callkernel(k, gx, 1, 1, bx, 1, 1);

fail:
    for (unsigned int i = 0; i < count; i++) {
        free(strs[i]);
    }
    return res;
}

static const char *cuda_error(void)
{
    return hipGetErrorString(hipPeekAtLastError());
}

compyte_buffer_ops cuda_ops = {cuda_alloc, cuda_free, cuda_move, cuda_read, cuda_write, cuda_memset, cuda_offset, cuda_newkernel, cuda_freekernel, cuda_setkernelarg, cuda_setkernelargbuf, cuda_callkernel, cuda_elemwise, cuda_error};

/*
  Local Variables:
  mode:c++
  c-basic-offset:4
  c-file-style:"stroustrup"
  c-file-offsets:((innamespace . 0)(inline-open . 0))
  indent-tabs-mode:nil
  fill-column:79
  End:
*/
// vim: filetype=cpp:expandtab:shiftwidth=4:tabstop=8:softtabstop=4:textwidth=79 :
