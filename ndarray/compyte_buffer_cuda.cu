#include <stdlib.h>

#include "compyte_buffer.h"

#define CNDA_THREAD_SYNC hipDeviceSynchronize()

struct _gpudata {
    char *ptr;
};

static gpudata *cuda_alloc(void *ctx /* IGNORED */, size_t size)
{
    /* ctx is ignored since it is implied from the context stack */
    hipError_t err;
    gpudata *res;
    res = (gpudata *)malloc(sizeof(*res));
    if (res == NULL) {
        return NULL;
    }
    err = hipMalloc(&res->ptr, size);
    if (hipSuccess != err) {
        return NULL;
    }
    return res;
}

static void cuda_free(gpudata *d) {
    hipFree(d->ptr);
    free(d);
}

static int cuda_move(gpudata *dst, gpudata *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst->ptr, src->ptr, sz, hipMemcpyDeviceToDevice);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_read(void *dst, gpudata *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst, src->ptr, sz, hipMemcpyDeviceToHost);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_write(gpudata *dst, void *src, size_t sz)
{
    hipError_t err;
    err = hipMemcpy(dst->ptr, src, sz, hipMemcpyHostToDevice);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_memset(gpudata *dst, int data, size_t bytes)
{
    hipError_t err;
    err = hipMemset(dst->ptr, data, bytes);
    CNDA_THREAD_SYNC;
    if (hipSuccess != err) {
        return GA_IMPL_ERROR;
    }
    return 0;
}

static int cuda_offset(gpudata *buf, int off) {
    buf->ptr += off;
    return 0;
}

static const char *cuda_error(void)
{
    return hipGetErrorString(hipPeekAtLastError());
}

compyte_buffer_ops cuda_ops = {cuda_alloc, cuda_free, cuda_move, cuda_read, cuda_write, cuda_memset, cuda_offset, cuda_error};

/*
  Local Variables:
  mode:c++
  c-basic-offset:4
  c-file-style:"stroustrup"
  c-file-offsets:((innamespace . 0)(inline-open . 0))
  indent-tabs-mode:nil
  fill-column:79
  End:
*/
// vim: filetype=cpp:expandtab:shiftwidth=4:tabstop=8:softtabstop=4:textwidth=79 :
